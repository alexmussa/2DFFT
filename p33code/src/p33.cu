#include "hip/hip_runtime.h"
//Allie Alexander I. Mussa - ECE 6122 - Final Project - Fall '18
//2D-DFT - Normal - CUDA Implementation (p33)
//Last Edit - 12/13/18 09:30 AM

#include <iostream>
#include <cmath>
#include <chrono>
#include "complex.h"
#include "input_image.h"
#define T_P_B 1024

int im_width = 0;
int im_height = 0;
int RowsPerThread = 0;
bool inverse;
 
Complex *im_data, *im_data_tps, *im_transformed;
Complex *Wnk;

using namespace std;

__global__ void DFT1D(Complex* data, Complex* Wnk,Complex* data_fft, const int im_height, const int im_width, const bool inverse){

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(index < im_width*im_height){
        data_fft[index] = 0;
        int factor = 1;

        if(im_width > blockDim.x){
            factor = im_width/blockDim.x;
            for(int i = 0; i < im_width; i++){
                *(data_fft + index) = *(data_fft + index) + (*(data + i + (blockIdx.x/factor)*im_width + int(threadIdx.x/im_width)*im_width) * *(Wnk + i + (index%im_width)*im_width));
            }
        }else{
            for(int i = 0; i < im_width; i++){
                *(data_fft + index) = *(data_fft + index) + (*(data + i + (blockIdx.x)*T_P_B + int(threadIdx.x/im_width)*im_width) * *(Wnk + i + (index%im_width)*im_width));
            }
        }

        

        if(inverse == true){
            *(data_fft + index) = *(data_fft + index) * float(1/float(im_width)); 
        }
    }
}

__global__ void transpose(Complex* data, Complex* data_transpose, int im_width, int im_height){
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index < im_width*im_height){
        *(data_transpose + index) = *(data + int(index/im_width) + (index%im_width)*im_width);
    }
}

int main(int argc, char **argv){
    auto t1 = chrono::high_resolution_clock::now();
    if(string(argv[1]) != string("forward") && string(argv[1]) != string("reverse")){
        printf("Error: expecting argument 1 to be 'forward' or 'reverse'. Instead, received %s. Exitting...", argv[1]);
        exit(1);
    }
 
    if(string(argv[1]) == "reverse"){
        inverse = true;
    }else if(string(argv[1]) == "forward"){
        inverse = false;
    }
 
    string filename = string(argv[2]);
    string outfilename = string(argv[3]);
    
    InputImage image(filename.c_str());

    im_width = image.get_width();
    im_height = image.get_height();
 
    im_data = new Complex [im_width * im_height];
    im_data = image.get_image_data();

    Complex *d_Wnk, *d_im_data, *d_im_data_ft;
    Complex *im_data_rec = new Complex [im_width*im_height];

    Wnk = new Complex [im_width*im_width];
    
    if(inverse == false){
        for(int n = 0; n < im_width; n++){
            for(int k = 0; k < im_width; k++){
                *(Wnk + n*im_width + k) = Complex(cos(2*M_PI*k*n/im_width), -1*sin(2*M_PI*k*n/im_width));
            }
        }
    }else{
        for(int n = 0; n < im_width; n++){
            for(int k = 0; k < im_width; k++){
                *(Wnk + n*im_width + k) = Complex(cos(2*M_PI*k*n/im_width), sin(2*M_PI*k*n/im_width));
            }
        }
    }

    int im_size = im_width * im_height;

    hipMalloc((void**)&d_Wnk, im_width*im_width*sizeof(Complex));
    hipMalloc((void**)&d_im_data, im_width*im_height*sizeof(Complex));
    hipMalloc((void**)&d_im_data_ft, im_width*im_height*sizeof(Complex));

    hipMemcpy(d_Wnk, Wnk, im_width*im_width*sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(d_im_data, im_data, im_width*im_height*sizeof(Complex), hipMemcpyHostToDevice);

    DFT1D<<<(im_size + T_P_B -1)/T_P_B,T_P_B>>>(d_im_data,d_Wnk,d_im_data_ft,im_height,im_width,inverse);
    hipDeviceSynchronize();
    transpose<<<(im_size + T_P_B -1)/T_P_B,T_P_B>>>(d_im_data_ft,d_im_data, im_width, im_height);
    hipDeviceSynchronize();
    DFT1D<<<(im_size + T_P_B -1)/T_P_B,T_P_B>>>(d_im_data,d_Wnk,d_im_data_ft,im_height,im_width,inverse);
    hipDeviceSynchronize();
    transpose<<<(im_size + T_P_B -1)/T_P_B,T_P_B>>>(d_im_data_ft,d_im_data, im_width, im_height);

    hipMemcpy(im_data_rec, d_im_data, im_width*im_height*sizeof(Complex), hipMemcpyDeviceToHost);
    
    if(inverse == false){
        image.save_image_data(outfilename.c_str(),im_data_rec,im_width,im_height);
    }else{
        image.save_image_data_real(outfilename.c_str(),im_data_rec,im_width,im_height);
    }
    
    hipFree(d_Wnk);
    hipFree(d_im_data);
    hipFree(d_im_data_ft);

    auto t2 = chrono::high_resolution_clock::now();
    chrono::duration<double> duration = (t2-t1);
    cout << duration.count() << '\n';
}